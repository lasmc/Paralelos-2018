/*Matrix-Vector*/

#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__
void matrixVectorKernel(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(i<n)
	{
		C[i] = 0;
		for(int j=0;j<n;j++)
			 C[i] += A[i*n+j] * B[j];

	}
	
}



void matrixVector(float* A, float* B, float* C, int tam)
{
	int sizeA = (tam*tam) * sizeof(float);
	int size =  tam * sizeof(float);
	float *d_A,*d_B,*d_C;

	hipMalloc((void**)&d_A,sizeA);
	hipMalloc((void**)&d_B,size);
	hipMalloc((void**)&d_C,size);

	hipMemcpy(d_A,A,sizeA,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

	matrixVectorKernel<<<ceil(tam/256.0),256>>>(d_A,d_B,d_C,tam);
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

	hipFree(d_A);hipFree(d_B);hipFree(d_C);
	
}

int main()
{
	int n = 3;
	float *h_A,*h_B,*h_C;
	h_A = new float[n*n];
	h_B = new float[n*n];
	h_C = new float[n*n];

	for(int i = 0; i < n; i++)
	{
	   for(int j = 0; j < n; j++)
	     h_A[i*n+j] = 2;
    }

    for(int i = 0; i < n; i++)
	{
	    h_B[i] = 3;
    }

    matrixVector(h_A,h_B,h_C,n);
    for(int i = 0; i < n; i++){
    	cout<<h_C[i]<<" ; ";
  	}

   cout<<endl;
   return 0;

	
}